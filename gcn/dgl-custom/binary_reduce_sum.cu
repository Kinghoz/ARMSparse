#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file kernel/cuda/binary_reduce_sum.cu
 * \brief CUDA kernels for binary reduce sum
 */
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"
#include "../utils.h"
#include "../csr_interface.h"

using minigun::advance::RuntimeConfig;

namespace dgl {
namespace kernel {
namespace cuda {
// specialization for cusparse

template <typename DType>
hipsparseStatus_t Xcsrmm2(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const DType* alpha, const hipsparseMatDescr_t descrA,
    const DType* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const DType* B, int ldb, const DType* beta, DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPSPARSE_STATUS_EXECUTION_FAILED;
}

template <>
hipsparseStatus_t Xcsrmm2<float>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const float* alpha, const hipsparseMatDescr_t descrA,
    const float* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const float* B, int ldb, const float* beta, float* C, int ldc) {
  return hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <>
hipsparseStatus_t Xcsrmm2<double>(hipsparseHandle_t handle, hipsparseOperation_t transA,
    hipsparseOperation_t transB, int m, int n, int k, int nnz,
    const double* alpha, const hipsparseMatDescr_t descrA,
    const double* csrValA, const int* csrRowPtrA, const int* csrColIndA,
    const double* B, int ldb, const double* beta, double* C, int ldc) {
  return hipsparseDcsrmm2(handle, transA, transB, m, n, k, nnz,
      alpha, descrA, csrValA, csrRowPtrA, csrColIndA,
      B, ldb, beta, C, ldc);
}

template <typename DType>
hipblasStatus_t Xgeam(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const DType* alpha, const DType* A, int lda,
    const DType* beta, const DType* B, int ldb,
    DType* C, int ldc) {
  LOG(INFO) << "Not supported dtype";
  return HIPBLAS_STATUS_EXECUTION_FAILED;
}

template <>
hipblasStatus_t Xgeam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const float* alpha, const float* A, int lda,
    const float* beta, const float* B, int ldb,
    float* C, int ldc) {
  return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <>
hipblasStatus_t Xgeam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
    hipblasOperation_t transb, int m, int n,
    const double* alpha, const double* A, int lda,
    const double* beta, const double* B, int ldb,
    double* C, int ldc) {
  return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda,
      beta, B, ldb, C, ldc);
}

template <typename DType>
void CusparseCsrmm2(
    const RuntimeConfig& rtcfg,
    const aten::CSRMatrix& csr,
    const DType* B_data, DType* C_data,
    int x_length) {
  // We use csrmm2 to perform following operation:
  // C = A x B, where A is a sparse matrix in csr format, B is the dense matrix for node
  // feature tensor. However, since cusparse only supports column-major, while our tensor
  // is stored in row-major, the actual computation is:
  // C = trans(A x trans(B)).
  // Currently, we use cublasXgeam to implement transposition and allocate intermediate
  // workspace memory for this.
  const int m = csr.num_rows;
  const int n = x_length;
  const int k = csr.num_cols;
  const int nnz = csr.indices->shape[0];
  const DType alpha = 1.0;
  const DType beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(rtcfg.ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, rtcfg.stream));
  // allocate matrix for temporary transposed output
  DType* trans_out = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, m * n * sizeof(DType)));
  // all one data array
  DType* valptr = static_cast<DType*>(device->AllocWorkspace(rtcfg.ctx, nnz * sizeof(DType)));
  utils::Fill<kDLGPU>(rtcfg.ctx, valptr, nnz, static_cast<DType>(1.));
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  CUSPARSE_CALL(Xcsrmm2<DType>(
      thr_entry->cusparse_handle,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      m, n, k, nnz, &alpha,
      descr, valptr,
      static_cast<int32_t*>(csr.indptr->data),
      static_cast<int32_t*>(csr.indices->data),
      B_data, n, &beta, trans_out, m));
  device->FreeWorkspace(rtcfg.ctx, valptr);
  // transpose the output matrix
  if (!thr_entry->cublas_handle) {
    CUBLAS_CALL(hipblasCreate(&(thr_entry->cublas_handle)));
  }
  CUBLAS_CALL(hipblasSetStream(thr_entry->cublas_handle, rtcfg.stream));
  CUBLAS_CALL(Xgeam<DType>(
      thr_entry->cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n, m,
      &alpha, trans_out, m,
      &beta, nullptr, n,
      C_data, n));
  device->FreeWorkspace(rtcfg.ctx, trans_out);
}


__device__ __forceinline__ float sum_reduce(float acc, float x) {
  return acc + x;
}

__device__ __forceinline__ float sum_init() {
  return 0;
}

__global__ void topoCacheCoarsenSPMMKernel(
  int m, int k, const int* A_indptr, const int* A_indices, const float* B, float* C
) {
  extern __shared__ int sh[];
  int sm_offset = (threadIdx.y<<5);
  int thread_idx = sm_offset+threadIdx.x;

  int rid = blockDim.y*blockIdx.x+threadIdx.y;
  if (rid<m) {

    int cid = (blockIdx.y<<6)+threadIdx.x;
    int lb = A_indptr[rid];
    int hb = A_indptr[rid+1];
    int ptr = lb+threadIdx.x;
    int offset;
    float acc1 = sum_init();
    float acc2 = sum_init();
    if (blockIdx.y != gridDim.y-1) {
      for (int jj=lb; jj<hb; jj+=32) {
        if (ptr<hb) {
          sh[thread_idx] = A_indices[ptr]*k;
          // sh[thread_idx] = __ldg(A_indices+ptr)*k;
        }
        __syncwarp();
        ptr += 32;
        for (int kk=0; kk<32&&jj+kk<hb; kk++) {
          offset = sh[(sm_offset+kk)] + cid;
          acc1 = sum_reduce(acc1, B[offset]);
          acc2 = sum_reduce(acc2, B[(offset+32)]);
          // acc1 = sum_reduce(acc1, __ldg(B+offset));
          // acc2 = sum_reduce(acc2, __ldg(B+offset+32));
        }
        __syncwarp();
      }
      offset = rid*k+cid;
      C[offset] = acc1;
      C[offset+32] = acc2;
    }
    else { // threadIdx.y==blockDim.y-1
      int nout = (k-cid+31)/32;
      for (int jj=lb; jj<hb; jj+=32) {
        if (ptr<hb) {
          sh[thread_idx] = A_indices[ptr]*k;
          // sh[thread_idx] = __ldg(A_indices+ptr)*k;
        }
        __syncwarp();
        ptr += 32;
        for (int kk=0; kk<32&&jj+kk<hb; kk++) {
          offset = sh[(sm_offset+kk)] + cid;
          if (nout>0) {
          acc1 = sum_reduce(acc1, B[offset]);}
          // acc1 = sum_reduce(acc1, __ldg(B+offset)); }
          if (nout>1) {
          acc2 = sum_reduce(acc2, B[(offset+32)]);}
          // acc2 = sum_reduce(acc2, __ldg(B+offset+32));}
        }
        __syncwarp();
      }
      offset = rid*k+cid;
      if (nout>0) {
      C[offset] = acc1;}
      if (nout>1) {
      C[offset+32] = acc2;}
    }
  }
} 

__global__ void topoCacheSPMMKernel(
  int m, int k, const int* A_indptr, const int* A_indices, const float* B, float* C 
) {
  extern __shared__ int sh[];
  int sm_offset = (threadIdx.y<<5);
  int thread_idx = sm_offset + threadIdx.x;
  
  int cid = (blockIdx.y<<5)+threadIdx.x;
  int rid = blockDim.y*blockIdx.x+threadIdx.y;
    
  if (rid<m) {
    int lb = A_indptr[rid];
    int hb = A_indptr[(rid+1)];
    int offset;
    int ptr = lb+threadIdx.x;
    float acc1 = sum_init();
    if (blockIdx.y != gridDim.y-1) {
      for (int jj=lb; jj<hb; jj+=32) {
        if (ptr<hb) {
          sh[thread_idx] = A_indices[ptr]*k;
          // sh[thread_idx] = __ldg(A_indices+ptr)*k;
        }
        __syncwarp();
        ptr += 32;
        for (int kk=0; kk<32&&jj+kk<hb; kk++) {
          offset = sh[sm_offset+kk]+cid;
          acc1 = sum_reduce(acc1, B[offset]);
          // acc1 = sum_reduce(acc1, __ldg(B+offset));
        }
        __syncwarp();
      }
      offset = rid*k+cid;
      C[offset] = acc1;
    }
    else { // threadIdx.y==blockDim.y-1
      int nout = (k-cid+31)/32;
      for (int jj=lb; jj<hb; jj+=32) {
        if (ptr<hb) {
          sh[thread_idx] = A_indices[ptr]*k;
          // sh[thread_idx] = __ldg(A_indices+ptr)*k;
        }
        __syncwarp();
        ptr += 32;
        for (int kk=0; kk<32&&jj+kk<hb; kk++) {
          offset = sh[(sm_offset+kk)] + cid;
          if (nout>0) {
          acc1 = sum_reduce(acc1, B[offset]);}
          // acc1 = sum_reduce(acc1, __ldg(B+offset)); }
        }
        __syncwarp();
      }
      offset = rid*k+cid;
      if (nout>0) {
      C[offset] = acc1;}
    }
  }
}

__global__ void topoSimpleSPMMKernel(
  int m, int k, const int* A_indptr, const int* A_indices, const float* B, float* C 
) {
  int rid = blockDim.y*blockIdx.x+threadIdx.y;
  if (rid<m) {
    int lb = A_indptr[rid];
    int hb = A_indptr[(rid+1)];
    float acc1 = sum_init();
    int offset;
    for (int ptr=lb; ptr<hb; ptr++) {
      // offset = __ldg(A_indices+ptr)*k+threadIdx.x;
      // acc1 = sum_reduce(acc1, __ldg(B+offset));
      offset = A_indices[ptr]*k+threadIdx.x;
      acc1 = sum_reduce(acc1, B[offset]);
    }
    C[(rid*k+threadIdx.x)] = acc1;
  }
}


template <typename DType>
int XTopoCsrmm(const RuntimeConfig& rtcfg,
  int m, int n,
  const int* A_indptr,
  const int* A_indices,
  const DType* B, DType* C) {
LOG(INFO) << "Not supported by custom spmm";
return -1;
}

template <>
int XTopoCsrmm<float>(const RuntimeConfig& rtcfg,
  int m, int n,
  const int* A_indptr,
  const int* A_indices,
  const float* B, float* C) {

  // LOG(INFO) << "Using custom spmm";
  if (n<32) {
    const int row_per_block = 128/n;
    const int n_block = (m+row_per_block-1)/row_per_block;
    topoSimpleSPMMKernel<<< dim3(n_block,1,1),dim3(n, row_per_block, 1), 0,rtcfg.stream>>>(m,n,A_indptr,A_indices,B,C);
    return 0;
  }
  if (n<64) {
    const int tile_k = (n+31)/32;
    const int n_block = (m+3)/4;
    topoCacheSPMMKernel<<< dim3(n_block,tile_k,1), dim3(32,4,1), 128*sizeof(int), rtcfg.stream>>>(m,n,A_indptr,A_indices,B,C);
    return 0;
  }
  else {
    const int tile_k = (n+63)/64;
    const int n_block = (m+8-1)/8;
    topoCacheCoarsenSPMMKernel<<< dim3(n_block,tile_k,1), dim3(32,8,1), 8*32*sizeof(int), rtcfg.stream>>>(m,n,A_indptr,A_indices,B,C);
    return 0;
  }
}

template <typename DType>
void CustomCsrmm(
  const RuntimeConfig& rtcfg,
  const aten::CSRMatrix& csr,
  const DType* B_data, DType* C_data,
  int x_length) {

  const int m = csr.num_rows;
  const int n = x_length;
  typedef int32_t Idx;

  int ret = XTopoCsrmm<DType> ( rtcfg,
    m, n, 
    static_cast<Idx*>(csr.indptr->data),
    static_cast<Idx*>(csr.indices->data),
    B_data, C_data
  );
  if (ret == -1) {
    CusparseCsrmm2(rtcfg, csr, B_data, C_data, x_length);
  }

  hipStreamSynchronize(rtcfg.stream);
  CUDA_CALL(hipGetLastError());
}

// forward

template <typename DType>
void FallbackCallBinaryReduce(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  typedef cuda::FunctorsTempl<Idx, DType, LeftSelector,
                        RightSelector, BinaryOp, Reducer>
          Functors;
  typedef cuda::BinaryReduce<Idx, DType, Functors> UDF;
  // csr
  auto outcsr = graph.GetOutCSRMatrix();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(outcsr.indptr, outcsr.indices);
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  if (RightSelector::target == binary_op::kEdge && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(outcsr.data->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, GData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

template <typename DType>
void FallbackCallBackwardBinaryReduce(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, DType>* gdata) {
  constexpr int XPU = kDLGPU;
  constexpr int Mode = binary_op::kGradLhs;
  typedef int32_t Idx;
  typedef SelectSrc LeftSelector;
  typedef SelectNone RightSelector;
  typedef BinaryUseLhs<DType> BinaryOp;
  typedef ReduceSum<kDLGPU, DType> Reducer;
  // For backward computation, we use reverse csr and switch dst and src.
  // This benefits the most common src_op_edge or copy_src case, because the
  // gradients of src are now aggregated into destination buffer to reduce
  // competition of atomic add.
  auto incsr = graph.GetInCSRMatrix();
  minigun::Csr<Idx> csr = utils::CreateCsr<Idx>(incsr.indptr, incsr.indices);
  typedef cuda::BackwardFunctorsTempl<Idx, DType,
          typename SwitchSrcDst<LeftSelector>::Type,
          typename SwitchSrcDst<RightSelector>::Type,
          BinaryOp, Reducer> Functors;
  typedef cuda::BackwardBinaryReduce<Mode, Idx, DType, Functors> UDF;
  // If the user-given mapping is none and the target is edge data, we need to
  // replace the mapping by the edge ids in the csr graph so that the edge
  // data is correctly read/written.
  if (LeftSelector::target == binary_op::kEdge
      && gdata->lhs_mapping == nullptr) {
    gdata->lhs_mapping = static_cast<Idx*>(incsr.data->data);
  }
  if (RightSelector::target == binary_op::kEdge
      && gdata->rhs_mapping == nullptr) {
    gdata->rhs_mapping = static_cast<Idx*>(incsr.data->data);
  }
  if (OutSelector<Reducer>::Type::target == binary_op::kEdge
      && gdata->out_mapping == nullptr) {
    gdata->out_mapping = static_cast<Idx*>(incsr.data->data);
  }
  // TODO(minjie): allocator
  minigun::advance::Advance<XPU, Idx, cuda::AdvanceConfig, BackwardGData<Idx, DType>, UDF>(
        rtcfg, csr, gdata, minigun::IntArray1D<Idx>());
}

}  // namespace cuda

template <>
void CallBinaryReduce<kDLGPU, int32_t, float, SelectSrc, SelectNone,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto csr = graph.GetInCSRMatrix();
    // cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        // gdata->x_length);
    cuda::CustomCsrmm(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->x_length);
  }
}

template <>
void CallBinaryReduce<kDLGPU, int32_t, double, SelectSrc, SelectNone,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    GData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    // cusparse use rev csr for csrmm
    auto csr = graph.GetInCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->lhs_data, gdata->out_data,
        gdata->x_length);
  }
}

// backward

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, float,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, float>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<float>(rtcfg, graph, gdata);
  } else {
    auto csr = graph.GetOutCSRMatrix();
    // cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        // gdata->x_length);
    cuda::CustomCsrmm(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->x_length);
  }
}

template <>
void CallBackwardBinaryReduce<kDLGPU, binary_op::kGradLhs, int32_t, double,
                              SelectSrc, SelectNone,
                              BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const CSRWrapper& graph,
    BackwardGData<int32_t, double>* gdata) {
  if (gdata->lhs_mapping || gdata->rhs_mapping || gdata->out_mapping) {
    cuda::FallbackCallBackwardBinaryReduce<double>(rtcfg, graph, gdata);
  } else {
    auto csr = graph.GetOutCSRMatrix();
    cuda::CusparseCsrmm2(rtcfg, csr, gdata->grad_out_data, gdata->grad_lhs_data,
        gdata->x_length);
  }
}

// generate definitions

#define REDUCER ReduceSum
#define XPU kDLGPU
#define IDX int32_t

EVAL(GEN_DTYPE, GEN_OP_TARGET, GEN_DEFINE);
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_OP_TARGET, GEN_BACKWARD_DEFINE);

}  // namespace kernel
}  // namespace dgl
